#include "hip/hip_runtime.h"

// sparse matrix in ELLPACK format
// no performance improvement



float elhm = 0.0f;
ty = threadIdx.x / bdx_mcs;
tx = threadIdx.x % bdx_mcs;


//#pragma unroll 4
for (int j = 0; j < mcs_nrow; j += bdy_mcs) { // y loop
    float elhm1 = 0.0f;
    int elhm2 = 0;

    const int m = j + ty;
    if (m < mcs_nrow) {
        for (int i = tx; i < mcs_ncol[m]; i += bdx_mcs) { // x loop
            const int l = mcs_ell->i[m][i];
            const float dx = lig_x2[l] - mcs_ell->x[m][i]; // do not use __LDG
            const float dy = lig_y2[l] - mcs_ell->y[m][i];
            const float dz = lig_z2[l] - mcs_ell->z[m][i];
            elhm1 += dx * dx + dy * dy + dz * dz;
            elhm2++; // never zero
        }
    }


    BlockReduceSum_2D_2_d_2 (bdy_mcs, bdx_mcs, elhm1, elhm2);

    if (threadIdx.x < bdy_mcs) {
        const int m = j + threadIdx.x;
        if (m < mcs_nrow)
            elhm += mcs_tcc[m] * sqrtf (elhm1 / (float) elhm2);
    }


} // lhm loop

WarpReduceSum_1_d_2 (elhm);
if (threadIdx.x == 0)
    e_s[7] = logf (elhm / mcs_nrow);


