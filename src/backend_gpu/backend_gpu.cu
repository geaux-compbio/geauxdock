#include "hip/hip_runtime.h"

#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <ctime>

#include <geauxdock.h>
#include <size.h>
#include <toggle.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <geauxdock.h>
#include <toggle.h>
#include <util_print.h>

#include <yeah/cuda/runtime/wrapper.h>
#include <yeah/cuda/runtime/event.h>
#include <yeah/cuda/runtime/info.h>
#include <yeah/cuda/kernel/util.h>
#include <yeah/c/timing.h>
#include <yeah/cpp/timer.hpp>



#define CUDAASSERT(condition, code) \
    if (!(condition)) printf("Assertion failure. Code %s%n\n", code)

#include "kernel_cuda_l2_reduce_notemplate.cu"
#include "kernel_cuda_l2_util.cu"
#include "kernel_cuda_l1_initcurand.cu"
#include "kernel_cuda_l1_montecarlo.cu"



// GPU timer can't measure memory copy
// CPU timer can't measure mc kernel


__global__
void
hello_d ()
{
}


void
InitCurand (hiprandState **s)
{
    srand (time (0));
    for (int g = 0; g < NGPU; ++g) {
        hipSetDevice (g);

        InitCurand_d <<< GD, BD >>> (s[g], rand () + g);
        CUDA_LAST_ERR ();
    }
}



void
Dock (Complex *ch,
    Record *rh,
    Complex **cd,
    Record **rd,
    ParaT **pt,
    hiprandState **curandstate_d)
{
    //printf ("runmc: begin %f\n", HostTimeNow ());
    yeah::Timer e[11];
    //yeah::cuda::EventSD e[11];


    //GetPrintCudaFuncArributes ((void (*)) MonteCarlo_d, "MonteCarlo_d");
    //GetPrintCudaFuncArributes2 ((void (*)) MonteCarlo_d, "MonteCarlo_d", GD, BD, 0);


    e[10].Start ();
    const int steps_total = ch->mcpara.steps_total;
    const int steps_per_dump = ch->mcpara.steps_per_dump;


    e[3].Start ();
    printf ("Start kernels\n");
    for (int g = 0; g < NGPU; ++g) {
        hipSetDevice (g);
        MonteCarlo_d <<< GD, BD >>> (cd[g], rd[g], 0, 1, curandstate_d[g]);
        CUDA_LAST_ERR ();
    }
    e[3].Stop ();



    for (int s1 = 0; s1 < steps_total; s1 += steps_per_dump) {
        printf ("\t%d / %d \n", s1, steps_total);

        e[4].Start ();
        for (int g = 0; g < NGPU; ++g) {
            hipSetDevice (g);
            MonteCarlo_d <<< GD, BD >>> (cd[g], rd[g], s1, steps_per_dump, curandstate_d[g]);
            CUDA_LAST_ERR ();
            //hipDeviceSynchronize();
        }

        //yeah::Timer eeee;
        //eeee.Start ();
        // copy ligand record from GPU to CPU memory
        // use synchronized copy to ensure multi-device consistency
        for (int g = 0; g < NGPU; ++g) {
            hipSetDevice (g);
            CUDA_ERR (hipMemcpy (rh + pt[g]->rep_begin, rd[g], pt[g]->record_sz, hipMemcpyDeviceToHost));
        }
        // eeee.Stop ();
        // printf ("launcher: time of memory copy D2H %f\n", eeee.Span());
        e[4].Stop ();
#include <kernel_dump.C>
    }

    Record *record = rh;
#include <kernel_print.C>


    e[10].Stop ();


#include <kernel_print_timer.C>
    //PrintSummary (ch);
#include <kernel_print_benchmark.C>


}

