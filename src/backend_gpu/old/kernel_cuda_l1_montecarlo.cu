#include "hip/hip_runtime.h"

#define MYRAND (MyRand_d(curandstate_d))

//#define MYRAND 0.54f


// templating s2max cause a slow down from 337 to 331



// python: sqrt_2_pi_inv = -1.0 / math.sqrt (3.1415926535897932384626433 * 2.0)
#define SQRT_2_PI_INV  -0.398942f



__global__ void
//__launch_bounds__(512, 1)
MonteCarlo_d (Complex * __restrict__ complex,
    Record * __restrict__ record,
    const int s1, const int s2max, hiprandState *curandstate_d)
{
    const int bidx = blockDim.x * threadIdx.y + threadIdx.x;

#if 1
    for (int r = complex->rep_begin + blockIdx.x; r <= complex->rep_end; r += gridDim.x) {

        // constant
        // pointers
        __shared__ ReplicaMC * __restrict__ rep;
        __shared__ Ligand * __restrict__ lig;
        __shared__ Protein * __restrict__ prt;
        __shared__ Psp * __restrict__ psp;
        __shared__ Kde * __restrict__ kde;
        __shared__ Mcs * __restrict__ mcs;
        __shared__ EnePara * __restrict__ enepara;


        // temporary write-read
        __shared__ float move_scale[6]; // translation x y z, rotation x y z
        __shared__ float movematrix[6]; // translation x y z, rotation x y z
        __shared__ float rot[3][3]; // rotz roty rotx
        __shared__ float lig_x1[MAXLIG];
        __shared__ float lig_y1[MAXLIG];
        __shared__ float lig_z1[MAXLIG];
        __shared__ float lig_x2[MAXLIG];
        __shared__ float lig_y2[MAXLIG];
        __shared__ float lig_z2[MAXLIG];


        // constant
        // ligand
        __shared__ int lig_t[MAXLIG];
        __shared__ float lig_c[MAXLIG];
        __shared__ float lig_center[3];


        // constant
        // protein
        __shared__ float prt_pocket_center[3];


        // constant
        // mcs
        __shared__ float mcs_tcc[MAX_MCS_ROW];


        // constant
        // enepara
        __shared__ float enepara_p1a[MAXTP2][MAXTP1];
        __shared__ float enepara_p2a[MAXTP2][MAXTP1];
        __shared__ float enepara_pmf0[MAXTP2][MAXTP1];
        __shared__ float enepara_pmf1[MAXTP2][MAXTP1];
        __shared__ float enepara_hdb0[MAXTP2][MAXTP1];
        __shared__ float enepara_hdb1[MAXTP2][MAXTP1];
        __shared__ float enepara_hpl0[MAXTP2];
        __shared__ float enepara_hpl1[MAXTP2];
        __shared__ float enepara_hpl2[MAXTP2];
        __shared__ float enepara_a_para[MAXWEI];
        __shared__ float enepara_b_para[MAXWEI];
        __shared__ float enepara_w[MAXWEI];
        __shared__ float enepara_lj0;
        __shared__ float enepara_lj1;
        __shared__ float enepara_el1;
        __shared__ float enepara_el0;
        __shared__ float enepara_a1;
        __shared__ float enepara_b1;
        __shared__ float enepara_kde2;
        __shared__ float enepara_kde3_inv;


        // constant
        // scalars
        __shared__ float sqrt_2_pi_inv;
        __shared__ int lig_natom, prt_npoint, kde_npoint, mcs_nrow;


        // temporary write-read vars
        __shared__ int is_accept_s;



        if (bidx == 0) {
            rep = &complex->replica[r];
            lig = &complex->lig[rep->idx_lig];
            prt = &complex->prt[rep->idx_prt];
            psp = &complex->psp;
            kde = &complex->kde;
            mcs = &complex->mcs[0];
            enepara = &complex->enepara;
        }

        __syncthreads ();


        if (bidx < 6)
            move_scale[bidx] = complex->mcpara.move_scale[bidx];

        for (int l = threadIdx.y; l < MAXTP2; l += blockDim.y) {
            for (int p = threadIdx.x; p < MAXTP1; p += blockDim.x) {
                enepara_p1a[l][p] = enepara->p1a[l][p];
                enepara_p2a[l][p] = enepara->p2a[l][p];
                enepara_pmf0[l][p] = enepara->pmf0[l][p];
                enepara_pmf1[l][p] = enepara->pmf1[l][p];
                enepara_hdb0[l][p] = enepara->hdb0[l][p];
                enepara_hdb1[l][p] = enepara->hdb1[l][p];
            }
        }

        for (int l = bidx; l < MAXTP2; l += blockDim.x * blockDim.y) {
            enepara_hpl0[l] = enepara->hpl0[l];
            enepara_hpl1[l] = enepara->hpl1[l];
            enepara_hpl2[l] = enepara->hpl2[l];
        }

        if (bidx < MAXWEI - 1) {
            enepara_a_para[bidx] = enepara->a_para[bidx];
            enepara_b_para[bidx] = enepara->b_para[bidx];
            enepara_w[bidx] = enepara->w[bidx];
        }

        if (bidx == 0) {
            enepara_lj0 = enepara->lj0;
            enepara_lj1 = enepara->lj1;
            enepara_el1 = enepara->el1;
            enepara_el0 = enepara->el0;
            enepara_a1 = enepara->a1;
            enepara_b1 = enepara->b1;
            enepara_kde2 = enepara->kde2;
            enepara_kde3_inv = 1.0f / enepara->kde3;

            sqrt_2_pi_inv = -1.0f / sqrtf (2.0f * PI);
            lig_natom = lig->lig_natom;
            prt_npoint = prt->prt_npoint;
            kde_npoint = complex->size.kde_npoint;
            mcs_nrow = complex->size.mcs_nrow;
            is_accept_s = rep->is_accept;

            record[r - complex->rep_begin].next_entry = 0; // reset the record's entry point
        }

        __syncthreads ();

        for (int l = bidx; l < lig_natom; l += blockDim.x * blockDim.y) {
            lig_t[l] = lig->t[l];
            lig_c[l] = lig->c[l];
        }

        if (bidx < 3) {
            lig_center[bidx] = lig->center[bidx];
            prt_pocket_center[bidx] = prt->pocket_center[bidx];
        }

        for (int m = bidx; m < mcs_nrow; m += blockDim.x * blockDim.y) {
            mcs_tcc[m] = mcs[m].tcc;
        }

        __syncthreads ();







        for (int s2 = 0; s2 < s2max; ++s2) {


            /////////////////////////////////////////////////////////////////////////////
            // record old states
            // 1.0% time
            if (bidx == 0 && is_accept_s == 1) {
                rep->step = s1 + s2;

                const int rr = r - complex->rep_begin;
                const int next_entry = record[rr].next_entry;
                record[rr].replica[next_entry] = *rep;
                record[rr].next_entry = next_entry + 1;
            }






            /////////////////////////////////////////////////////////////////////////////
            // move

            if (bidx < 6) {

#if IS_AWAY == 0
                const float fixed_var = 0.0f;
#elif IS_AWAY == 1
                const float fixed_var = 44.5f;
#endif

#if 1
                float moveamount;
                if (s2max != 1)
                    moveamount = MYRAND;
                else
                    moveamount = fixed_var;
#endif

#if 0
                float moveamount = (s2max != 1) ? MYRAND : fixed_var;
#endif




                movematrix[bidx] = move_scale[bidx] * moveamount + rep->movematrix[bidx];
            }

            __syncthreads ();


            if (bidx == 0) {
                // http://en.wikipedia.org/wiki/Euler_angles
                // http://upload.wikimedia.org/math/e/9/c/e9cf817bce9c1780216921cd93233459.png
                // http://upload.wikimedia.org/math/f/4/e/f4e55dc2c9581007648967d29b15121e.png
                const float sin1 = sinf (movematrix[3]);
                const float cos1 = cosf (movematrix[3]);
                const float sin2 = sinf (movematrix[4]);
                const float cos2 = cosf (movematrix[4]);
                const float sin3 = sinf (movematrix[5]);
                const float cos3 = cosf (movematrix[5]);
                rot[0][0] = cos1 * cos2;
                rot[0][1] = cos1 * sin2 * sin3 - cos3 * sin1;
                rot[0][2] = sin1 * sin3 + cos1 * cos3 * sin2;
                rot[1][0] = cos2 * sin1;
                rot[1][1] = cos1 * cos3 + sin1 * sin2 * sin3;
                rot[1][2] = cos3 * sin1 * sin2 - cos1 * sin3;
                rot[2][0] = -1 * sin2;
                rot[2][1] = cos2 * sin3;
                rot[2][2] = cos2 * cos3;
            }

            __syncthreads ();

            // rotation, translation, coordinate system transformation
            for (int l = bidx; l < lig_natom; l += blockDim.x * blockDim.y) {
                const float x1 = lig->x[l];
                const float y1 = lig->y[l];
                const float z1 = lig->z[l];
                lig_x1[l] = rot[0][0] * x1 + rot[0][1] * y1 + rot[0][2] * z1 + movematrix[0] + lig_center[0];
                lig_y1[l] = rot[1][0] * x1 + rot[1][1] * y1 + rot[1][2] * z1 + movematrix[1] + lig_center[1];
                lig_z1[l] = rot[2][0] * x1 + rot[2][1] * y1 + rot[2][2] * z1 + movematrix[2] + lig_center[2];

                const float x2 = lig->x2[l];
                const float y2 = lig->y2[l];
                const float z2 = lig->z2[l];
                lig_x2[l] = rot[0][0] * x2 + rot[0][1] * y2 + rot[0][2] * z2 + movematrix[0] + lig_center[0];
                lig_y2[l] = rot[1][0] * x2 + rot[1][1] * y2 + rot[1][2] * z2 + movematrix[1] + lig_center[1];
                lig_z2[l] = rot[2][0] * x2 + rot[2][1] * y2 + rot[2][2] * z2 + movematrix[2] + lig_center[2];
            }

            __syncthreads ();





            /////////////////////////////////////////////////////////////////////////////
            // calcenergy

            float evdw = 0.0f;
            float eele = 0.0f;
            float epmf = 0.0f;
            float epsp = 0.0f;
            float ehdb = 0.0f;
            float ehpc = 0.0f;


#if CALC_PRT == 1

            // lig loop, ~30
            //#pragma unroll 8
            for (int j = 0; j < lig_natom; j += blockDim.y) {
                float ehpc1 = 0.0f;

                {
                    const int l = j + threadIdx.y;
                    if (l < lig_natom) {
                        const int lig__t = lig_t[l];

                        //#pragma unroll 4
                        // prt loop, ~300
                        for (int p = threadIdx.x; p < prt_npoint; p += blockDim.x) {
                            const int prt__t = CUDA_LDG_D (prt->t[p]);

                            const float dx = lig_x1[l] - CUDA_LDG_D (prt->x[p]);
                            const float dy = lig_y1[l] - CUDA_LDG_D (prt->y[p]);
                            const float dz = lig_z1[l] - CUDA_LDG_D (prt->z[p]);
                            const float dst_pow2 = dx * dx + dy * dy + dz * dz;
                            const float dst_pow4 = dst_pow2 * dst_pow2;
                            const float dst = sqrtf (dst_pow2);


                            /* hydrophobic potential */
                            {
                                // worse
#if 0
                                if (CUDA_LDG_D (prt->cdc[p]) == 1 && dst_pow2 <= 81.0f OROR1) {
                                    ehpc1 += CUDA_LDG_D (prt->hpp[p]) *
                                        (1.0f - (3.5f / 81.0f * dst_pow2 -
                                                 4.5f / 81.0f / 81.0f * dst_pow4 +
                                                 2.5f / 81.0f / 81.0f / 81.0f * dst_pow4 * dst_pow2 -
                                                 0.5f / 81.0f / 81.0f / 81.0f / 81.0f * dst_pow4 * dst_pow4));
                                }
#endif
                                // better
#if 1
                                const int mask_hpc = (CUDA_LDG_D (prt->cdc[p]) == 1 && dst_pow2 <= 81.0f OROR1);
                                const float temp_hpc = CUDA_LDG_D (prt->hpp[p]) *
                                    (1.0f - (3.5f / 81.0f * dst_pow2 -
                                             4.5f / 81.0f / 81.0f * dst_pow4 +
                                             2.5f / 81.0f / 81.0f / 81.0f * dst_pow4 * dst_pow2 -
                                             0.5f / 81.0f / 81.0f / 81.0f / 81.0f * dst_pow4 * dst_pow4));
                                ehpc1 += temp_hpc * mask_hpc;
#endif
                            }





#if 1
                            /* L-J potential */
                            // p1a[MAXTP2][MAXTP1]
                            // p2a[MAXTP2][MAXTP1]
                            {
                                const float p1 = enepara_p1a[lig__t][prt__t] / (dst_pow4 * dst_pow4 * dst);
                                const float p2 = enepara_p2a[lig__t][prt__t] / (dst_pow4 * dst_pow2);
                                const float p4 = p1 * enepara_lj0 * (1.0f + enepara_lj1 * dst_pow2) + 1.0f;
                                evdw += (p1 - p2) / p4;
                            }
#endif




                            /* electrostatic potential */
                            {
                                // better
#if 1
                                const float s1 = enepara_el1 * dst;
                                float g1;
                                if (s1 < 1.0f OROR1)
                                    g1 = enepara_el0 + enepara_a1 * s1 * s1 + enepara_b1 * s1 * s1 * s1;
                                else
                                    g1 = 1.0f / s1;
                                eele += lig_c[l] * CUDA_LDG_D (prt->ele[p]) * g1;
#endif
                                // worse
#if 0
                                const float s1 = enepara_el1 * dst;
                                const float g1_0 = enepara_el0 + enepara_a1 * s1 * s1 + enepara_b1 * s1 * s1 * s1;
                                const float g1_1 = 1.0f / s1;
                                const float g1 = (s1 < 1) ? g1_0 : g1_1;
                                eele += lig_c[l] * CUDA_LDG_D (prt->ele[p]) * g1;
#endif
                            }



                            /* contact potential */
                            // pmf0[MAXTP2][MAXTP1]
                            // pmf1[MAXTP2][MAXTP1]
                            // psp[MAXTP2][MAXPRO]

                            const float dst_minus_pmf0 = dst - enepara_pmf0[lig__t][prt__t];

                            epmf += enepara_pmf1[lig__t][prt__t] / (1.0f + expf ((-0.5f * dst + 6.0f) * dst_minus_pmf0));



                            /* pocket-specific potential */
                            // the senmatics do not match with the original program:
                            // if (found psp[][])
                            //   accumulate to epsp
                            // else
                            //   do nothing
                            {
                                // better
#if 1
                                if (CUDA_LDG_D (prt->c[p]) == 2 && dst_minus_pmf0 <= 0 OROR1) {
                                    const int i1 = CUDA_LDG_D (prt->seq3r[p]);
                                    epsp += CUDA_LDG_D (psp->psp[lig__t][i1]); // sparse matrix, indirect dereference
                                    // performance profiling:
                                    //epsp += float (lig__t + i1); // improve from 336 to 352, not worth doing

                                }
#endif
                                // worse
#if 0
                                const int mask_psp = (CUDA_LDG_D (prt->c[p]) == 2 && dst_minus_pmf0 <= 0 OROR1);
                                const int i1 = CUDA_LDG_D (prt->seq3r[p]);
                                const float temp_psp = CUDA_LDG_D (psp->psp[lig__t][i1]); // sparse matrix, indirect dereference
                                epsp += temp_psp * mask_psp;
#endif
                            }




                            /* hydrogen bond potential */
                            // hdb0[MAXTP2][MAXTP1]
                            // hdb1[MAXTP2][MAXTP1]
                            {
                                // better
#if 1
                                const float hdb0 = enepara_hdb0[lig__t][prt__t];
                                if (hdb0 > 0.1f OROR1) {
                                    const float hdb1 = enepara_hdb1[lig__t][prt__t];
                                    const float hdb3 = (dst - hdb0) * hdb1;
                                    ehdb += hdb1 * expf (-0.5f * hdb3 * hdb3);
                                }
#endif
                                // worse
#if 0
                                const float hdb0 = enepara_hdb0[lig__t][prt__t];
                                const int mask_hdb = (hdb0 > 0.1f OROR1);
                                const float hdb1 = enepara_hdb1[lig__t][prt__t];
                                const float hdb3 = (dst - hdb0) * hdb1;
                                const float temp_hdb = hdb1 * expf (-0.5f * hdb3 * hdb3);
                                ehdb += temp_hdb * mask_hdb;
#endif
                            }


                        } // prt loop
                    } // if (l < lig_natom)
                } // end of scope "const int l = j + threadIdx.y"



                /* hydrophobic restraits*/
                // hpl0[MAXTP2]
                // hpl1[MAXTP2]
                // hpl2[MAXTP2]

                BlockReduceSum_2D_1_d <float> (bidx, ehpc1);
                if (threadIdx.y == 0 && threadIdx.x < blockDim.y) {
                    const int l = j + threadIdx.x;
                    if (l < lig_natom) {
                        const int lig__t = lig_t[l];
                        const float hpc2 = (ehpc1 - enepara_hpl0[lig__t]) / enepara_hpl1[lig__t]; // div hpl is faster than mul hpl_inv
                        ehpc += 0.5f * hpc2 * hpc2 - enepara_hpl2[lig__t];
                    }
                }



            } // lig loop


            BlockReduceSum_5_d <float, float, float, float, float> (bidx, evdw, eele, epmf, epsp, ehdb);
            WarpReduceSum_1_d <float> (ehpc);

#endif

            __shared__ float e[MAXWEI];
            if (bidx == 0) {
                e[0] = evdw; // 0 - vdw 
                e[1] = eele; // 1 - ele
                e[2] = epmf; // 2 - pmf (CP)
                e[3] = epsp; // 3 - psp (PS CP)
                e[4] = ehdb * SQRT_2_PI_INV; // 4 - hdb (HB)
                //e[4] = ehdb * sqrt_2_pi_inv; // 4 - hdb (HB)
                e[5] = ehpc; // 5 - hpc (HP)
            }








#if CALC_KDE == 1
            /* kde potential */
            // fully optimized

            float ekde = 0.0f;

            // lig loop, ~30
            for (int j = 0; j < lig_natom; j += blockDim.y) {
                float ekde1 = 0.0f;
                int ekde2 = 0;

                {
                    const int l = j + threadIdx.y;
                    if (l < lig_natom) {

                        //#pragma unroll 2
                        // kde loop, ~400
                        for (int k = threadIdx.x; k < kde_npoint; k += blockDim.x) {
#if 1
                            if (lig_t[l] == kde->t[k] OROR1) {
                                const float dx = lig_x1[l] - kde->x[k];
                                const float dy = lig_y1[l] - kde->y[k];
                                const float dz = lig_z1[l] - kde->z[k];
                                const float kde_dst_pow2 = dx * dx + dy * dy + dz * dz;
                                ekde1 += expf (enepara_kde2 * kde_dst_pow2);
                                ekde2++;
#endif

#if 0
                                const int mask_kde = (lig_t[l] == kde->t[k] OROR1);
                                const float dx = lig_x1[l] - kde->x[k];
                                const float dy = lig_y1[l] - kde->y[k];
                                const float dz = lig_z1[l] - kde->z[k];
                                const float kde_dst_pow2 = dx * dx + dy * dy + dz * dz;
                                const float temp_kde = expf (enepara_kde2 * kde_dst_pow2);
                                ekde1 += temp_kde * mask_kde;
                                ekde2 += mask_kde;
#endif


                            }
                        } // kde loop
                    } // if (l < lig_natom)
                }

                BlockReduceSum_2D_2_d <float, int> (bidx, ekde1, ekde2);
                if (threadIdx.y == 0 && threadIdx.x < blockDim.y) {
                    const int l = j + threadIdx.x;
                    if (l < lig_natom && ekde2 != 0)
                        ekde += ekde1 / (float) ekde2;
                }

            } // lig loop

            WarpReduceSum_1_d <float> (ekde);
            if (bidx == 0)
                e[6] = ekde * enepara_kde3_inv;

#endif


#if CALC_MCS == 1
            /* position restraints */
            // fully optimized

            float elhm = 0.0f;

            // lhm loop, ~11
            // unrolling outer loop does not help
            //#pragma unroll 4
            for (int j = 0; j < mcs_nrow; j += blockDim.y) {
                float elhm1 = 0.0f;
                int elhm2 = 0;

                {
                    const int m = j + threadIdx.y;
                    if (m < mcs_nrow) {
                        // lig loop, ~30

                        for (int l = threadIdx.x; l < lig_natom; l += blockDim.x) {
                            if (CUDA_LDG_D (mcs[m].x[l + 1]) != MCS_INVALID_COORD OROR1) {
                                const float dx = lig_x2[l] - CUDA_LDG_D (mcs[m].x[l + 1]);
                                const float dy = lig_y2[l] - CUDA_LDG_D (mcs[m].y[l + 1]);
                                const float dz = lig_z2[l] - CUDA_LDG_D (mcs[m].z[l + 1]);
                                elhm1 += dx * dx + dy * dy + dz * dz;
                                elhm2++;
                            }
                        } // lig loop
                    } // if (m < mcs_nrow)
                }

                BlockReduceSum_2D_2_d <float, int> (bidx, elhm1, elhm2);
                if (threadIdx.y == 0 && threadIdx.x < blockDim.y) {
                    const int m = j + threadIdx.x;
                    if (m < mcs_nrow && elhm2 != 0)
                        elhm += mcs_tcc[m] * sqrtf (elhm1 / (float) elhm2);
                }

            } // lhm loop

            WarpReduceSum_1_d <float> (elhm);
            if (bidx == 0)
                e[7] = logf (elhm / mcs_nrow);

#endif


#if CALC_DST == 1
            // fully optimized

            {
                float dst;
                if (bidx < 3) {
                    dst = lig_center[bidx] + movematrix[bidx] - prt_pocket_center[bidx];
                    dst = dst * dst;
                }
                if (bidx == 0) {
                    dst += __shfl (dst, 1) + __shfl (dst, 2);
                    e[8] = sqrtf (dst);
                }
            }
#endif


            __syncthreads ();

            // normalization
            if (bidx < 7)
                e[bidx] = e[bidx] / lig_natom;
            if (bidx < MAXWEI - 1)
                e[bidx] = enepara_a_para[bidx] * e[bidx] + enepara_b_para[bidx];
            float etotal = 0.0f;
            if (bidx < MAXWEI - 1)
                etotal = enepara->w[bidx] * e[bidx]; // enepara->w is faster than enepara_w
            WarpReduceSum_1_d <float> (etotal);
            if (bidx == 0)
                e[MAXWEI - 1] = etotal;



            __syncthreads ();


            ////////////////////////////////////////////////////////////////////////
            // accept

            if (bidx == 0) {
                const float delta_energy = e[MAXWEI - 1] - rep->energy[MAXWEI -1];
                const float beta = complex->temp[rep->idx_tmp].minus_beta;
#if 1
                float rand;
                if (s2max != 1)
                    rand = MYRAND;
                else
                    rand = 0.0f; // force to accept if s2max == 1
#endif
#if 0
                float rand = (s2max != 1) ? MYRAND : 0.0f;
#endif

                is_accept_s = (rand < expf (delta_energy * beta));  // mybeta < 0
            }
            __syncthreads ();
            if (is_accept_s == 1) {
                if (bidx < MAXWEI)
                    rep->energy[bidx] = e[bidx];
                if (bidx < 6)
                    rep->movematrix[bidx] = movematrix[bidx];
            }


        } // s2 loop



        if (bidx == 0)
            rep->is_accept = is_accept_s;

    } // replica loop
#endif
}

