#include "hip/hip_runtime.h"
/*
#include <cstdlib>
#include <cstdio>

#include "geauxdock.h"
#include "gpu.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
 */



// warp_id == bidx / warpSize == bidx >> WARPshift
// warp_land == bidx % warpSize == bidx & WARPmask
#ifndef WARPshift
#define WARPshift 5
#endif
#ifndef WARPmask
#define WARPmask 0b11111
#endif







template <typename T0, typename T1, typename T2, typename T3, typename T4>
__inline__ __device__
static void
BlockReduceSum_5_d (T0 &a0, T1 &a1, T2 &a2, T3 &a3, T4 &a4)
{

    const int bidx = threadIdx.x;

    __shared__ T0 a0s[WARPperB];
    __shared__ T1 a1s[WARPperB];
    __shared__ T2 a2s[WARPperB];
    __shared__ T3 a3s[WARPperB];
    __shared__ T4 a4s[WARPperB];

    // 1st level warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
        a1 += __shfl_xor (a1, stride);
        a2 += __shfl_xor (a2, stride);
        a3 += __shfl_xor (a3, stride);
        a4 += __shfl_xor (a4, stride);
    }

    {
        const int warp_lane = bidx & WARPmask;
        const int warp_id = bidx >> WARPshift;
        if (warp_lane == 0) {
            a0s[warp_id] = a0;
            a1s[warp_id] = a1;
            a2s[warp_id] = a2;
            a3s[warp_id] = a3;
            a4s[warp_id] = a4;
        }
    }

    __syncthreads ();
    a0 = (bidx < WARPperB) ? a0s[bidx] : 0;
    a1 = (bidx < WARPperB) ? a1s[bidx] : 0;
    a2 = (bidx < WARPperB) ? a2s[bidx] : 0;
    a3 = (bidx < WARPperB) ? a3s[bidx] : 0;
    a4 = (bidx < WARPperB) ? a4s[bidx] : 0;

    // 2nd level warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
        a1 += __shfl_xor (a1, stride);
        a2 += __shfl_xor (a2, stride);
        a3 += __shfl_xor (a3, stride);
        a4 += __shfl_xor (a4, stride);
    }
}









template <typename T0>
__inline__ __device__
static void
WarpReduceSum_1_d (T0 &a0)
{
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
    }
}






#if 0

template <typename T0, typename T1>
__inline__ __device__
static void
BlockReduceSum_2D_2_d (const int bdy, const int bdx, T0 &a0, T1 &a1)
{
    const int bidx = threadIdx.x;

    __shared__ T0 a0s[WARPperB];
    __shared__ T1 a1s[WARPperB];
    const int warp_lane = bidx & WARPmask;


    // warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
        a1 += __shfl_xor (a1, stride);
    }

    {
        const int warp_id = bidx >> WARPshift;
        if (warp_lane == 0) {
            a0s[warp_id] = a0;
            a1s[warp_id] = a1;
        }
    }

    __syncthreads ();

    const int warp_x_per_b = bdx >> WARPshift;

    if (bidx < bdy) {
        a0 = 0;
        a1 = 0;
#pragma unroll
        for (int s = 0; s < warp_x_per_b; ++s) {
            a0 += a0s[warp_x_per_b * bidx + s];
            a1 += a1s[warp_x_per_b * bidx + s];
        }
    }


}

#endif







template <typename T0>
__inline__ __device__
static void
BlockReduceSum_2D_1_d (const int bdy, const int bdx, T0 &a0)
{
    const int bidx = threadIdx.x;

    __shared__ T0 a0s[WARPperB];
    const int warp_lane = bidx & WARPmask;


    // warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
    }

    {
        const int warp_id = bidx >> WARPshift;
        if (warp_lane == 0) {
            a0s[warp_id] = a0;
        }
    }

    __syncthreads ();

    const int warp_x_per_b = bdx >> WARPshift;

    if (bidx < bdy) {
        a0 = 0;
#pragma unroll
        for (int s = 0; s < warp_x_per_b; ++s) {
            a0 += a0s[warp_x_per_b * bidx + s];
        }
    }

}
