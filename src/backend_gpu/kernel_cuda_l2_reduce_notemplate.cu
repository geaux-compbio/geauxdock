
#include <hip/hip_runtime.h>
/*
#include <cstdlib>
#include <cstdio>

#include "geauxdock.h"
#include "gpu.cuh"

#include <cuda.h>
#include <curand.h>
#include <curand_kernel.h>
 */



// warp_id == bidx / warpSize == bidx >> WARPshift
// warp_land == bidx % warpSize == bidx & WARPmask

#ifndef WARPshift
#define WARPshift 5
#endif
#ifndef WARPmask
#define WARPmask 0b11111
#endif


#define WARPperB_MAX 32




__inline__ __device__
static void
BlockReduceSum_5_d_2 (float &a0, float &a1, float &a2, float &a3, float &a4)
{

    const int bidx = threadIdx.x;

    __shared__ float a0s[WARPperB_MAX];
    __shared__ float a1s[WARPperB_MAX];
    __shared__ float a2s[WARPperB_MAX];
    __shared__ float a3s[WARPperB_MAX];
    __shared__ float a4s[WARPperB_MAX];

    // 1st level warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
        a1 += __shfl_xor (a1, stride);
        a2 += __shfl_xor (a2, stride);
        a3 += __shfl_xor (a3, stride);
        a4 += __shfl_xor (a4, stride);
    }

    {
        const int warp_lane = bidx & WARPmask;
        const int warp_id = bidx >> WARPshift;
        if (warp_lane == 0) {
            a0s[warp_id] = a0;
            a1s[warp_id] = a1;
            a2s[warp_id] = a2;
            a3s[warp_id] = a3;
            a4s[warp_id] = a4;
        }
    }

    __syncthreads ();
    const int cond = (bidx * warpSize) < blockDim.x;
    a0 = cond ? a0s[bidx] : 0.0f;
    a1 = cond ? a1s[bidx] : 0.0f;
    a2 = cond ? a2s[bidx] : 0.0f;
    a3 = cond ? a3s[bidx] : 0.0f;
    a4 = cond ? a4s[bidx] : 0.0f;

    // 2nd level warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
        a1 += __shfl_xor (a1, stride);
        a2 += __shfl_xor (a2, stride);
        a3 += __shfl_xor (a3, stride);
        a4 += __shfl_xor (a4, stride);
    }
}


__inline__ __device__
static void
BlockReduceSum_1_d_2 (float &a0)
{

    const int bidx = threadIdx.x;

    __shared__ float a0s[WARPperB_MAX];

    // 1st level warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1)
        a0 += __shfl_xor (a0, stride);

    {
        const int warp_lane = bidx & WARPmask;
        const int warp_id = bidx >> WARPshift;
        if (warp_lane == 0)
            a0s[warp_id] = a0;
    }

    __syncthreads ();
    const int cond = (bidx * warpSize) < blockDim.x;
    a0 = cond ? a0s[bidx] : 0.0f;

    // 2nd level warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1)
        a0 += __shfl_xor (a0, stride);
}









__inline__ __device__
static void
WarpReduceSum_1_d_2 (float &a0)
{
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
    }
}









__inline__ __device__
static void
BlockReduceSum_2D_2_d_2 (const int bdy, const int bdx, float &a0, int &a1)
{
    const int bidx = threadIdx.x;

    __shared__ float a0s[WARPperB_MAX];
    __shared__ int a1s[WARPperB_MAX];
    const int warp_lane = bidx & WARPmask;


    // warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
        a1 += __shfl_xor (a1, stride);
    }

    {
        const int warp_id = bidx >> WARPshift;
        if (warp_lane == 0) {
            a0s[warp_id] = a0;
            a1s[warp_id] = a1;
        }
    }

    __syncthreads ();

    const int warp_x_per_b = bdx >> WARPshift;

    if (bidx < bdy) {
        a0 = 0.0f;
        a1 = 0;
#pragma unroll
        for (int s = 0; s < warp_x_per_b; ++s) {
            a0 += a0s[warp_x_per_b * bidx + s];
            a1 += a1s[warp_x_per_b * bidx + s];
        }
    }


}








__inline__ __device__
static void
BlockReduceSum_2D_1_d_2 (const int bdy, const int bdx, float &a0)
{
    const int bidx = threadIdx.x;

    __shared__ float a0s[WARPperB_MAX];
    const int warp_lane = bidx & WARPmask;


    // warp reduction
#pragma unroll
    for (int stride = 16; stride > 0; stride >>= 1) {
        a0 += __shfl_xor (a0, stride);
    }

    {
        const int warp_id = bidx >> WARPshift;
        if (warp_lane == 0) {
            a0s[warp_id] = a0;
        }
    }

    __syncthreads ();

    const int warp_x_per_b = bdx >> WARPshift;

    if (bidx < bdy) {
        a0 = 0;
#pragma unroll
        for (int s = 0; s < warp_x_per_b; ++s) {
            a0 += a0s[warp_x_per_b * bidx + s];
        }
    }


}
